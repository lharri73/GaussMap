#include "hip/hip_runtime.h"
/*****************************************************************************
 * This contains the implementation of many different classes and the device *
 * functions used in various kernels. Keeping the implementation in one file *
 * allows the linker to perform link time optimization. Although they can be *
 * seperated, it yeilds a major drawback on performance at runtime.          *
 ****************************************************************************/
#include <sstream>
#include <hip/hip_math_constants.h>     // HIP_PI_F
#include "gaussMap.cuh"

__device__
size_t array_index(size_t row, size_t col, array_info *info){
    // helper function to find the array index
    return (row * info->cols) + col;
}

__device__
Position indexDiff(size_t row, size_t col, RadarData_t *radarData, size_t radarPointIdx, 
                   array_info *radarInfo, array_info *mapInfo, array_rel *mapRel){
    // Calculate the position of the cell at (row,col) relative to the radar point at 
    // radarPointIdx
    Position pos = index_to_position(row, col, mapInfo, mapRel);
    
    float rPosx = radarData[array_index(threadIdx.x, 0, radarInfo)];
    float rPosy = radarData[array_index(threadIdx.x, 1, radarInfo)];
    // printf("rpos %d x: %f, y: %f\n", threadIdx.x, rPosx, rPosy);

    Position difference(
        pos.x - rPosx,
        pos.y - rPosy
    );
    return difference;
}

__device__ 
Position index_to_position(size_t row, size_t col, array_info *info, array_rel *relation){
    // find the position from center of map given cell index
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = (col - center_x);
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    Position ret(
        x_offset / (float)relation->res,
        y_offset / (float)relation->res
    );
    return ret;
}

__device__
float calcPdf(float stdDev, float mean, float radius){
    // calculate the pdf of the given radar point based on the radius
    float variance = pow(stdDev, 2);
    float first = (1 / stdDev) * rsqrt(2 * HIP_PI_F);
    float exponent = -1 * pow(radius - mean, 2) / (2 * variance);
    float second = exp(exponent);
    return first*second;
}

__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      RadarData_t *radarData, 
                      array_info *mapInfo, 
                      array_rel* mapRel, 
                      array_info* radarInfo,
                      float* distributionInfo){
                          
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            // find where the cell is relative to the radar point
            Position diff = indexDiff(row, col, 
                                      radarData, threadIdx.x, 
                                      radarInfo, mapInfo, mapRel);
            // don't calculate the pdf of this cell if it's too far away
            if(diff.radius > distributionInfo[2])
                continue;

            float pdfVal = calcPdf(distributionInfo[0], distributionInfo[1], diff.radius);
            // printf("pdf: %f\n", pdfVal);
            atomicAdd(&gaussMap[array_index(row,col,mapInfo)], pdfVal);
        }
    }
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed

    checkCudaError(hipMalloc(&mapInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&radarInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&mapRel_cuda, sizeof(struct Array_Relationship)));
    checkCudaError(hipMalloc(&radarDistri_c, 3*sizeof(float)));
    
    checkCudaError(hipMemcpy(mapInfo_cuda, &mapInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarInfo_cuda, &radarInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(mapRel_cuda, &mapRel, sizeof(struct Array_Relationship), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarDistri_c, radarDistri, 3*sizeof(float), hipMemcpyHostToDevice));

    // dispatch the kernel with `numPoints` threads
    radarPointKernel<<<1,radarInfo.rows>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c
    );

    // wait untill all threads sync
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

}

//-----------------------------------------------------------------------------
// Camera point stuff

__global__ 
void camPointKernel(mapType_t* gaussMap, 
                    float *camData, 
                    array_info *mapInfo, 
                    array_rel* mapRel, 
                    array_info* camInfo,
                    float* distributionInfo,
                    camVal_t *camClassVals,
                    array_info* camClassInfo){
    /*
    For every camera point, go through each cell in the map and determine the PDF
    value iff the cell is within the cutoff radius. If it is, add the value of 
    the PDF to the gaussMap. This function also records the point's PDF value and
    class in the camClassVals array. This includes the pdf and the class with the
    pdf as the most significant 32 bits of the 64 bit element size array. We use 
    a union to get this value as an unsigned long long int for an atomicMax that
    will store the maximum of either the value in the array or the value passed
    as input (what is calculated for this particular camera point). This allows us
    to keep a map of classes based on camera points, keeping only the class data 
    originating from the closest camera point if overlap occurs. 
    */
                          
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            // find where the cell is relative to the radar point
            Position diff = indexDiff(row, col, 
                                      camData, threadIdx.x, 
                                      camInfo, mapInfo, mapRel);
            // don't calculate the pdf of this cell if it's too far away
            if(diff.radius > distributionInfo[2])
                continue;

            float pdfVal = calcPdf(distributionInfo[0], distributionInfo[1], diff.radius);
            atomicAdd(&gaussMap[array_index(row,col,mapInfo)], pdfVal);

            union {
                camVal_t camVal;
                unsigned long long int ulong;
            } cat;

            cat.ulong = 0; // initialize to 0
            cat.camVal.probability = pdfVal;
            cat.camVal.classVal = (uint32_t)camData[array_index(threadIdx.x, 2, camInfo)];
            
            atomicMax((unsigned long long*)&camClassVals[array_index(row,col,camClassInfo)], cat.ulong);
        }
    }
}

void GaussMap::calcCameraMap(){
    camClassInfo.rows = mapInfo.rows;
    camClassInfo.cols = mapInfo.cols;
    camClassInfo.elementSize = sizeof(struct CamVal);

    checkCudaError(hipMalloc(&cameraInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&camClassInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&cameraDistri_c, 3*sizeof(float)));

    checkCudaError(hipMemcpy(cameraInfo_cuda, &cameraInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(camClassInfo_cuda, &camClassInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(cameraDistri_c, cameraDistri, 3*sizeof(float), hipMemcpyHostToDevice));

    // allocate the camera class array
    checkCudaError(hipMalloc(&cameraClassData, camClassInfo.elementSize * camClassInfo.rows * camClassInfo.cols));
    checkCudaError(hipMemset(cameraClassData, 0, camClassInfo.elementSize * camClassInfo.rows * camClassInfo.cols));

    camPointKernel<<<1,cameraInfo.rows>>>(
        array,
        cameraData,
        mapInfo_cuda,
        mapRel_cuda,
        cameraInfo_cuda,
        cameraDistri_c,
        cameraClassData,
        camClassInfo_cuda
    );

    // wait untill all threads sync
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "camPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }
}

//-----------------------------------------------------------------------------
// maxima locating

__global__
void calcMaxKernel(uint8_t *isMax, float* array, 
                   array_info *mapInfo){
    int row = threadIdx.x;
    int col = blockIdx.x;
    if(row == 0 || row == mapInfo->rows) return;
    if(col == 0 || col == mapInfo->cols) return;
    
    float curVal = array[array_index(row,col, mapInfo)];
    if(curVal == 0) return; // not a max if it's zero

    for(int i = -1; i <= 1; i++){
        for(int j = -1; j <= 1; j++){
            if(array[array_index(row+i, col+j, mapInfo)] > curVal)
                return;
        }
    }

    isMax[array_index(row,col,mapInfo)] = 1;

}

std::vector<uint16_t> GaussMap::calcMax(){
    uint8_t *isMax_cuda;
    checkCudaError(hipMalloc(&isMax_cuda, sizeof(uint8_t) * mapInfo.rows * mapInfo.cols));

    // initialize isMax to 0
    checkCudaError(hipMemset(isMax_cuda, 0, sizeof(uint8_t) * mapInfo.rows * mapInfo.cols));

    dim3 maxGridSize(mapInfo.rows, 1, 1);
    dim3 maxBlockSize(mapInfo.cols, 1, 1);

    calcMaxKernel<<<maxGridSize, maxBlockSize>>>(
        isMax_cuda,
        array,
        mapInfo_cuda
    );

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // copy back to host so we can iterate over it
    uint8_t *isMax = (uint8_t*)calloc(sizeof(uint8_t), mapInfo.rows * mapInfo.cols);
    checkCudaError(hipMemcpy(isMax, isMax_cuda, sizeof(uint8_t) * mapInfo.rows * mapInfo.cols, hipMemcpyDeviceToHost));
    
    // now we don't need the device memory since it's on the host
    checkCudaError(hipFree(isMax_cuda));

    std::vector<uint16_t> ret;   // stored as (row,col,row,col,row,col,...)
    for(uint16_t row = 0; row < mapInfo.rows; row++){
        for(uint16_t col = 0; col < mapInfo.cols; col++){
            if(isMax[(size_t)(row * mapInfo.cols + col)] == 1){
                ret.push_back(row);
                ret.push_back(col);
            }
        }
    }
    return ret;
}

//-----------------------------------------------------------------------------
// Position class implementation
__device__ 
Position::Position(float X, float Y) : x(X), y(Y){
    radius = hypotf(x,y);
}

__device__
Position::Position(){

}

__device__
void Position::recalc(){
    radius = hypotf(x,y);
}

//-----------------------------------------------------------------------------
// 