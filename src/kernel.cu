#include "hip/hip_runtime.h"
/*****************************************************************************
 * This contains the implementation of many different classes and the device *
 * functions used in various kernels. Keeping the implementation in one file *
 * allows the linker to perform link time optimization. Although they can be *
 * seperated, it yeilds a major drawback on performance at runtime.          *
 ****************************************************************************/
#include <sstream>
#include <hip/hip_math_constants.h>     // HIP_PI_F
#include "gaussMap.cuh"
#include <iostream>

__device__ __forceinline__
size_t array_index(size_t row, size_t col, const array_info *info){
    // helper function to find the array index
    return (row * info->cols) + col;
}

__device__
Position indexDiff(size_t row, size_t col, const RadarData_t *radarData, size_t radarPointIdx, 
                   const array_info *radarInfo, const array_info *mapInfo, const array_rel *mapRel){
    // Calculate the position of the cell at (row,col) relative to the radar point at 
    // radarPointIdx
    Position pos = index_to_position(row, col, mapInfo, mapRel);
    
    float rPosx = radarData[array_index(radarPointIdx, 0, radarInfo)];
    float rPosy = radarData[array_index(radarPointIdx, 1, radarInfo)];
    // printf("rpos %d x: %f, y: %f\n", threadIdx.x, rPosx, rPosy);

    Position difference(
        pos.x - rPosx,
        pos.y - rPosy
    );
    return difference;
}

__device__ 
Position index_to_position(size_t row, size_t col, const array_info *info, const array_rel *relation){
    // find the position from center of map given cell index
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = (col - center_x);
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    Position ret(
        x_offset / (float)relation->res,
        y_offset / (float)relation->res
    );
    return ret;
}

__device__ __forceinline__
float calcPdf(float stdDev, float mean, float radius){
    // calculate the pdf of the given radar point based on the radius
    float variance = pow(stdDev, 2);
    float first = (1 / stdDev) * rsqrt(2 * HIP_PI_F);
    float exponent = -1 * pow(radius - mean, 2) / (2 * variance);
    float second = exp(exponent);
    return first*second;
}

__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      const RadarData_t *radarData, 
                      const array_info *mapInfo, 
                      const array_rel* mapRel, 
                      const array_info* radarInfo,
                      const distInfo_t* distributionInfo,
                      radarId_t *radarIds){
    // In this function, the radar point id is threadIdx.x

    union{
        radarId_t radData;
        unsigned long long int ulong;
    } un;

    for(size_t col = 0; col < mapInfo->cols; col++){
        // find where the cell is relative to the radar point
        Position diff = indexDiff(blockIdx.x, col, 
                                    radarData, threadIdx.x, 
                                    radarInfo, mapInfo, mapRel);
        // don't calculate the pdf of this cell if it's too far away
        if(diff.radius > distributionInfo->distCutoff)
            continue;

        float pdfVal = calcPdf(distributionInfo->stdDev, distributionInfo->mean, diff.radius);
        // printf("pdf: %f\n", pdfVal);
        atomicAdd(&gaussMap[array_index(blockIdx.x,col,mapInfo)], pdfVal);

        un.radData.radarId = threadIdx.x;
        un.radData.probability = pdfVal;
        atomicMax((unsigned long long int*)&radarIds[array_index(blockIdx.x, col, mapInfo)], un.ulong);

    }
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed
    checkCudaError(hipMemcpy(radarInfo_cuda, &radarInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));

    // dispatch the kernel with `numPoints x mapInfo.rows` threads
    radarPointKernel<<<mapInfo.rows,radarInfo.rows>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c,
        radarIds
    );
    
    // wait untill all threads sync
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

}

//-----------------------------------------------------------------------------
// maxima locating

__global__
void calcMaxKernel(maxVal_t *isMax, 
                  const float* array, 
                  const array_info *mapInfo,
                  const radarId_t *radarIds){
    int row = threadIdx.x;
    int col = blockIdx.x;
    if(row == 0 || row == mapInfo->rows) return;
    if(col == 0 || col == mapInfo->cols) return;
    
    float curVal = array[array_index(row,col, mapInfo)];
    if(curVal == 0) return; // not a max if it's zero

    maxVal_t *toInsert;
    toInsert = &isMax[array_index(row,col,mapInfo)];
    size_t iterator = 0;
    for(int i = -3; i <= 3; i++){
        for(int j = -3; j <= 3; j++){
            if(array[array_index(row+i, col+j, mapInfo)] > curVal)
                return;
            if(row+i >= 0 && col+j >= 0)
                toInsert->radars[iterator++] = radarIds[array_index(row+i, col+j, mapInfo)].radarId;
        }
    }

    toInsert->isMax = 1;
    toInsert->classVal = 0;
}

__device__ __forceinline__
float calcMean(size_t col, 
               const int16_t* radars, 
               const RadarData_t *radarData, 
               const array_info *radarInfo)
{
    float total = 0;
    size_t numPoints = 0;
    for(size_t i = 0; i < 49; i++){
        if(radars[i] == -1) continue;

        total += radarData[array_index(radars[i], col, radarInfo)];
        numPoints++;
    }
    
    return (total/numPoints);
}

__global__ 
void aggregateMax(const mapType_t *array, 
                  const array_info *mapInfo, 
                  const array_rel *mapRel,
                  const maxVal_t *isMax, 
                  float* ret, 
                  const radarId_t *radarIds,
                  const array_info* maxInfo, 
                  float minCutoff,
                  const RadarData_t *radarData, 
                  const array_info *radarInfo)
{
    // creates an array with the return information in the form of:
    // [row, col, class, pdfVal, vx, vy]
    size_t maxFound = 0;
    maxVal_t tmp;
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            tmp = isMax[(size_t)(row * mapInfo->cols + col)];
            if(tmp.isMax == 1 && array[row * mapInfo->cols + col] >= minCutoff){
                if(maxFound++ == threadIdx.x){
                    ret[array_index(threadIdx.x, 0, maxInfo)] = ((float)(row - mapInfo->rows/2.0) * -1.0) / mapRel->res;
                    ret[array_index(threadIdx.x, 1, maxInfo)] = (col - mapInfo->cols/2.0) / mapRel->res;
                    ret[array_index(threadIdx.x, 2, maxInfo)] = radarData[array_index(tmp.radars[49/2],3, radarInfo)];
                    ret[array_index(threadIdx.x, 3, maxInfo)] = array[row * mapInfo->cols + col];
                    ret[array_index(threadIdx.x, 4, maxInfo)] = calcMean(8, tmp.radars, radarData, radarInfo);
                    ret[array_index(threadIdx.x, 5, maxInfo)] = calcMean(9, tmp.radars, radarData, radarInfo);
                }
            }
        }
    }
}

std::pair<array_info,float*> GaussMap::calcMax(){
    maxVal_t *isMax_cuda;
    checkCudaError(hipMalloc(&isMax_cuda, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols));

    // initialize isMax to 0
    checkCudaError(hipMemset(isMax_cuda, 0, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols));

    dim3 maxGridSize(mapInfo.rows, 1, 1);
    dim3 maxBlockSize(mapInfo.cols, 1, 1);

    calcMaxKernel<<<maxGridSize, maxBlockSize>>>(
        isMax_cuda,
        array,
        mapInfo_cuda,
        radarIds
    );

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // copy back to host so we can iterate over it
    maxVal_t *isMax = (maxVal_t*)calloc(sizeof(maxVal_t), mapInfo.rows * mapInfo.cols);
    checkCudaError(hipMemcpy(isMax, isMax_cuda, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols, hipMemcpyDeviceToHost));
    
    float *arrayTmp = (float*)calloc(sizeof(float), mapInfo.rows * mapInfo.cols);
    checkCudaError(hipMemcpy(arrayTmp, array, sizeof(float) * mapInfo.rows * mapInfo.cols, hipMemcpyDeviceToHost));

    // find the number of maxima
    // this can be optimized later
    size_t numMax = 0;
    maxVal_t tmp;
    for(size_t row = 0; row < mapInfo.rows; row++){
        for(size_t col = 0; col < mapInfo.cols; col++){
            tmp = isMax[(size_t)(row * mapInfo.cols + col)];
            if(tmp.isMax == 1 && arrayTmp[row * mapInfo.cols + col] >= minCutoff){
                numMax++;
            }
        }
    }
    
    array_info maxData;
    maxData.cols = 6;
    maxData.rows = numMax;
    maxData.elementSize = sizeof(float);

   
    array_info *maxData_c;
    checkCudaError(hipMalloc(&maxData_c, sizeof(array_info)));
    checkCudaError(hipMemcpy(maxData_c, &maxData, sizeof(array_info), hipMemcpyHostToDevice));

    float *ret, *ret_c;
    checkCudaError(hipMalloc(&ret_c, sizeof(float) * maxData.rows * maxData.cols));

    aggregateMax<<<1, numMax>>>(
        array,
        mapInfo_cuda,
        mapRel_cuda,
        isMax_cuda,
        ret_c,
        radarIds,
        maxData_c,
        minCutoff,
        radarData,
        radarInfo_cuda
    );

    hipDeviceSynchronize();
    hipError_t error2 = hipGetLastError();
    if(error2 != hipSuccess){
        std::stringstream ss;
        ss << "aggregateMaxKernel launch failed\n";
        ss << hipGetErrorString(error2);
        throw std::runtime_error(ss.str());
    }

    ret = (float*)malloc(maxData.size());
    checkCudaError(hipMemcpy(ret, ret_c, maxData.size(), hipMemcpyDeviceToHost));

    safeCudaFree(ret_c);
    safeCudaFree(isMax_cuda);
    safeCudaFree(maxData_c);

    return std::pair<array_info,float*>(maxData,ret);
}

//-----------------------------------------------------------------------------
// Association kernel

__global__
void associateCameraKernel(
    const RadarData_t *radarData,
    const array_info *radarInfo,
    const float* camData,
    const array_info *camInfo,
    float* results,
    const array_info *resultInfo
){
    /*
    radarData: [row, col, class, pdfVal, vx, vy]
    cameraData: [x,y,class]
    */
    extern __shared__ float spaceMap[];
    array_info spaceMapInfo;
    
    spaceMapInfo.rows = radarInfo->rows;
    spaceMapInfo.cols = camInfo->rows;
    spaceMapInfo.elementSize = sizeof(float);

    int row = threadIdx.x;
    int col = threadIdx.y;

    float camX, camY;
    float radX, radY;
    camX = camData[array_index(col, 0, camInfo)];
    camY = camData[array_index(col, 1, camInfo)];

    radX = radarData[array_index(row, 0, radarInfo)];
    radY = radarData[array_index(row, 1, radarInfo)];

    float distance = hypotf(camX-radX, camY-radY);
    spaceMap[array_index(row,col,&spaceMapInfo)] = distance;
    printf("%f\n", distance);
    __syncthreads();
}

std::pair<array_info,float*> GaussMap::associateCamera(){
    // calculate the radar's maxima
    std::pair<array_info,float*> maxima = calcMax();
    array_info *maximaInfo;
    checkCudaError(hipMalloc(&maximaInfo, sizeof(array_info)));
    checkCudaError(hipMemcpy(maximaInfo, &(maxima.first), hipMemcpyHostToDevice));

    array_info assocInfo, *assocInfo_c;
    assocInfo.rows = maxima.first.rows + camInfo.rows;
    assocInfo.cols = 6; // [x,y,vx,vy,class,isValid]
    assocInfo.elementSize = sizeof(float);
    
    float* associated;
    checkCudaError(hipMalloc(&associated, assocInfo.size()));
    checkCudaError(hipMemset(associated, 0, assocInfo.size()));
    checkCudaError(hipMalloc(&assocInfo_c, sizeof(array_info)));
    checkCudaError(hipMemcpy(assocInfo_c, &assocInfo, sizeof(array_info), hipMemcpyHostToDevice));

    dim3 threadInfo(maxima.first.rows, camInfo.rows);

    associateCameraKernel<<<1, threadInfo, radarInfo.rows*camInfo.rows*sizeof(float)>>>(
        maxima.second,
        maximaInfo,
        camData,
        camInfo_cuda,
        associated,
        assocInfo_c
    );

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "associateCameraKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }
    
    return std::pair<array_info,float*>(assocInfo,associated);
}

//-----------------------------------------------------------------------------
// Position class implementation
__device__ 
Position::Position(float X, float Y) : x(X), y(Y){
    radius = hypotf(x,y);
}

__device__
Position::Position(){

}

__device__
void Position::recalc(){
    radius = hypotf(x,y);
}

//-----------------------------------------------------------------------------
