#include "hip/hip_runtime.h"
/*****************************************************************************
 * This contains the implementation of many different classes and the device *
 * functions used in various kernels. Keeping the implementation in one file *
 * allows the linker to perform link time optimization. Although they can be *
 * seperated, it yeilds a major drawback on performance at runtime.          *
 ****************************************************************************/
#include <sstream>
#include <hip/hip_math_constants.h>     // HIP_PI_F
#include "gaussMap.cuh"

__device__
size_t array_index(size_t row, size_t col, array_info *info){
    // helper function to find the array index
    return row * info->cols + col;
}

__device__
Position indexDiff(size_t row, size_t col, RadarData_t *radarData, size_t radarPointIdx, 
                   array_info *radarInfo, array_info *mapInfo, array_rel *mapRel){
    // Calculate the position of the cell at (row,col) relative to the radar point at 
    // radarPointIdx
    Position pos = index_to_position(row, col, mapInfo, mapRel);
    
    float rPosx = radarData[array_index(threadIdx.x, 0, radarInfo)];
    float rPosy = radarData[array_index(threadIdx.x, 1, radarInfo)];
    // printf("rpos %d x: %f, y: %f\n", threadIdx.x, rPosx, rPosy);

    Position difference(
        pos.x - rPosx,
        pos.y - rPosy
    );
    return difference;
}

__device__ 
Position index_to_position(size_t row, size_t col, array_info *info, array_rel *relation){
    // find the position from center of map given cell index
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = (col - center_x);
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    Position ret(
        x_offset / (float)relation->res,
        y_offset / (float)relation->res
    );
    return ret;
}

__device__
float calcPdf(float stdDev, float mean, float radius){
    // calculate the pdf of the given radar point based on the radius
    float variance = pow(stdDev, 2);
    float first = (1 / stdDev) * rsqrt(2 * HIP_PI_F);
    float exponent = -1 * pow(radius - mean, 2) / (2 * variance);
    float second = exp(exponent);
    return first*second;
}

__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      RadarData_t *radarData, 
                      array_info *mapInfo, 
                      array_rel* mapRel, 
                      array_info* radarInfo,
                      float* distributionInfo){
                          
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            // find where the cell is relative to the radar point
            Position diff = indexDiff(row, col, 
                                      radarData, threadIdx.x, 
                                      radarInfo, mapInfo, mapRel);
            // don't calculate the pdf of this cell if it's too far away
            if(diff.radius > distributionInfo[2])
                continue;

            float pdfVal = calcPdf(distributionInfo[0], distributionInfo[1], diff.radius);
            // printf("pdf: %f\n", pdfVal);
            atomicAdd(&gaussMap[array_index(row,col,mapInfo)], pdfVal);
        }
    }
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed

    checkCudaError(hipMalloc(&mapInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&radarInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&mapRel_cuda, sizeof(struct Array_Relationship)));
    checkCudaError(hipMalloc(&radarDistri_c, 3*sizeof(float)));
    
    checkCudaError(hipMemcpy(mapInfo_cuda, &mapInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarInfo_cuda, &radarInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(mapRel_cuda, &mapRel, sizeof(struct Array_Relationship), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarDistri_c, radarDistri, 3*sizeof(float), hipMemcpyHostToDevice));

    // dispatch the kernel with `numPoints` threads
    radarPointKernel<<<1,numPoints>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c
    );

    // wait untill all threads sync
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

}

//-----------------------------------------------------------------------------
// Derivative code implementation
__global__
void calcDerivativeKernel(float* f, array_info *fInfo, float* fPrime, array_info *fPrimeInfo){
    // https://en.wikipedia.org/wiki/Finite_difference#Multivariate_finite_differences 
    // ^ 5th equation
    size_t row,col;
    row = threadIdx.x +1;
    col = blockIdx.x +1;
    float first, second, third, fourth;
    first = f[array_index(row+1, col+1, fInfo)];
    second = f[array_index(row+1, col-1, fInfo)];
    third = f[array_index(row-1, col+1, fInfo)];
    fourth = f[array_index(row-1, col-1, fInfo)];

    fPrime[array_index(threadIdx.x, blockIdx.x, fPrimeInfo)] = (first - second - third + fourth) / 4.0;
    // printf("%d %d %d | %d %d %d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z);
}

void GaussMap::calcDerivative(){
    primeInfo.rows = mapInfo.rows -2;
    primeInfo.cols = mapInfo.cols -2;
    primeInfo.elementSize = sizeof(float);

    checkCudaError(hipMalloc(&primeInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMemcpy(primeInfo_cuda, &primeInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    if(mapInfo_cuda == nullptr)
        throw std::runtime_error("radar data must be added before the derivative can be calculated.");
    
    checkCudaError(hipMalloc(&arrayPrime, sizeof(float) * primeInfo.rows * primeInfo.cols));

    // dispatch the kernel with a single thread per cell
    dim3 primeGridSize(primeInfo.rows);
    dim3 primeBlockSize(primeInfo.cols);
    calcDerivativeKernel<<<primeGridSize,primeBlockSize>>>(
        array,
        mapInfo_cuda,
        arrayPrime,
        primeInfo_cuda
    );
    
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // now do it again...
    primePrimeInfo.rows = primeInfo.rows -2;
    primePrimeInfo.cols = primeInfo.cols -2;
    primePrimeInfo.elementSize = sizeof(float);

    checkCudaError(hipMalloc(&primePrimeInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMemcpy(primePrimeInfo_cuda, &primePrimeInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMalloc(&arrayPrimePrime, sizeof(float) * primePrimeInfo.rows * primePrimeInfo.cols));

    dim3 primePrimeGridSize(primePrimeInfo.rows);
    dim3 primePrimeBlockSize(primePrimeInfo.cols);
    calcDerivativeKernel<<<primePrimeGridSize,primePrimeBlockSize>>>(
        arrayPrime,
        primeInfo_cuda,
        arrayPrimePrime,
        primePrimeInfo_cuda
    );
    
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

}

//-----------------------------------------------------------------------------
// maxima locating

__global__
void calcMaxKernel(uint8_t *isMax, float* arrayPrime, 
                   array_info *primeInfo, float* arrayPrimePrime, 
                   array_info *primePrimeInfo){

}

std::vector<uint16_t> GaussMap::calcMax(){
    if(!arrayPrime || !arrayPrimePrime) // make sure these are allocated
        throw std::runtime_error("Derivative must be calculated before maxima can be located");

    uint8_t *isMax_cuda;
    checkCudaError(hipMalloc(&isMax_cuda, sizeof(uint8_t) * primePrimeInfo.rows * primePrimeInfo.cols));

    dim3 maxGridSize(primePrimeInfo.rows);
    dim3 maxBlockSize(primePrimeInfo.cols);

    calcMaxKernel<<<maxGridSize, maxBlockSize>>>(
        isMax_cuda,
        arrayPrime,
        primeInfo_cuda,
        arrayPrimePrime,
        primePrimeInfo_cuda
    );

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // copy back to host so we can iterate over it
    uint8_t *isMax = (uint8_t*)calloc(sizeof(uint8_t), primePrimeInfo.rows * primePrimeInfo.cols);
    checkCudaError(hipMemcpy(isMax, isMax_cuda, sizeof(uint8_t) * primePrimeInfo.rows * primePrimeInfo.cols, hipMemcpyDeviceToHost));
    
    // now we don't need the device memory since it's on the host
    checkCudaError(hipFree(isMax_cuda));

    std::vector<uint16_t> ret;   // stored as (row,col,row,col,row,col,...)
    for(uint16_t row = 0; row < primePrimeInfo.rows; row++){
        for(uint16_t col = 0; col < primePrimeInfo.cols; col++){
            if(isMax[(size_t)(row * primePrimeInfo.cols + col)]){
                ret.push_back(row);
                ret.push_back(col);
            }
        }
    }
    return ret;
}

//-----------------------------------------------------------------------------
// Position class implementation
__device__ 
Position::Position(float X, float Y) : x(X), y(Y){
    radius = hypotf(x,y);
}

__device__
Position::Position(){

}

__device__
void Position::recalc(){
    radius = hypotf(x,y);
}
