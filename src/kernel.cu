#include "hip/hip_runtime.h"
/*****************************************************************************
 * This contains the implementation of many different classes and the device *
 * functions used in various kernels. Keeping the implementation in one file *
 * allows the linker to perform link time optimization. Although they can be *
 * seperated, it yeilds a major drawback on performance at runtime.          *
 ****************************************************************************/
#include <sstream>
#include <hip/hip_math_constants.h>     // HIP_PI_F
#include "gaussMap.cuh"

__device__
size_t array_index(size_t row, size_t col, array_info *info){
    // helper function to find the array index
    return row * info->cols + col;
}

__device__
Position indexDiff(size_t row, size_t col, RadarData_t *radarData, size_t radarPointIdx, 
                   array_info *radarInfo, array_info *mapInfo, array_rel *mapRel){
    // Calculate the position of the cell at (row,col) relative to the radar point at 
    // radarPointIdx
    Position pos = index_to_position(row, col, mapInfo, mapRel);
    
    float rPosx = radarData[array_index(threadIdx.x, 0, radarInfo)];
    float rPosy = radarData[array_index(threadIdx.x, 1, radarInfo)];
    // printf("rpos %d x: %f, y: %f\n", threadIdx.x, rPosx, rPosy);

    Position difference(
        pos.x - rPosx,
        pos.y - rPosy
    );
    return difference;
}

__device__ 
Position index_to_position(size_t row, size_t col, array_info *info, array_rel *relation){
    // find the position from center of map given cell index
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = (col - center_x);
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    Position ret(
        x_offset / (float)relation->res,
        y_offset / (float)relation->res
    );
    return ret;
}

__device__
float calcPdf(float stdDev, float mean, float radius){
    // calculate the pdf of the given radar point based on the radius
    float variance = pow(stdDev, 2);
    float first = (1 / stdDev) * rsqrt(2 * HIP_PI_F);
    float exponent = -1 * pow(radius - mean, 2) / (2 * variance);
    float second = exp(exponent);
    return first*second;
}

__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      RadarData_t *radarData, 
                      array_info *mapInfo, 
                      array_rel* mapRel, 
                      array_info* radarInfo,
                      float* distributionInfo){
                          
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            // find where the cell is relative to the radar point
            Position diff = indexDiff(row, col, 
                                      radarData, threadIdx.x, 
                                      radarInfo, mapInfo, mapRel);
            // don't calculate the pdf of this cell if it's too far away
            if(diff.radius > distributionInfo[2])
                continue;

            float pdfVal = calcPdf(distributionInfo[0], distributionInfo[1], diff.radius);
            // printf("pdf: %f\n", pdfVal);
            atomicAdd(&gaussMap[array_index(row,col,mapInfo)], pdfVal);
        }
    }
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed

    checkCudaError(hipMalloc(&mapInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&radarInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&mapRel_cuda, sizeof(struct Array_Relationship)));
    checkCudaError(hipMalloc(&radarDistri_c, 3*sizeof(float)));
    
    checkCudaError(hipMemcpy(mapInfo_cuda, &mapInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarInfo_cuda, &radarInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(mapRel_cuda, &mapRel, sizeof(struct Array_Relationship), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarDistri_c, radarDistri, 3*sizeof(float), hipMemcpyHostToDevice));

    // dispatch the kernel with `numPoints` threads
    radarPointKernel<<<1,numPoints>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c
    );

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // wait untill all threads sync
    hipDeviceSynchronize();
}

//-----------------------------------------------------------------------------
// Derivative code implementation
__global__
void calcDerivativeKernel(float* f, array_info *fInfo, float* fprime, array_info *fPrimeInfo){
    // https://en.wikipedia.org/wiki/Finite_difference#Multivariate_finite_differences 
    // ^ 5th equation

}

float* GaussMap::calcDerivative(){
    primeInfo.rows = mapInfo.rows -2;
    primeInfo.cols = mapInfo.cols -2;
    primeInfo.elementSize = sizeof(float);

    checkCudaError(hipMalloc(&primeInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMemcpy(primeInfo_cuda, &primeInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    if(mapInfo_cuda == nullptr)
        throw std::runtime_error("radar data must be added before the derivative can be calculated.");
    
    checkCudaError(hipMalloc(&arrayPrime, sizeof(float) * primeInfo.rows * primeInfo.cols));

    // dispatch the kernel with a single thread per cell
    calcDerivativeKernel<<<primeInfo.rows, primeInfo.cols>>>(
        array,
        mapInfo_cuda,
        arrayPrime,
        primeInfo_cuda
    );

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    hipDeviceSynchronize();

    return nullptr;
}

//-----------------------------------------------------------------------------
// Position class implementation
__device__ 
Position::Position(float X, float Y) : x(X), y(Y){
    radius = hypotf(x,y);
}

__device__
Position::Position(){

}

__device__
void Position::recalc(){
    radius = hypotf(x,y);
}