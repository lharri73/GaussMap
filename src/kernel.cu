#include "hip/hip_runtime.h"
/*****************************************************************************
 * This contains the implementation of many different classes and the device *
 * functions used in various kernels. Keeping the implementation in one file *
 * allows the linker to perform link time optimization. Although they can be *
 * seperated, it yeilds a major drawback on performance at runtime.          *
 ****************************************************************************/
#include <sstream>
#include <hip/hip_math_constants.h>     // HIP_PI_F
#include "gaussMap.cuh"
#include <iostream>

__device__ __forceinline__
size_t array_index(size_t row, size_t col, array_info *info){
    // helper function to find the array index
    return (row * info->cols) + col;
}

__device__
Position indexDiff(size_t row, size_t col, RadarData_t *radarData, size_t radarPointIdx, 
                   array_info *radarInfo, array_info *mapInfo, array_rel *mapRel){
    // Calculate the position of the cell at (row,col) relative to the radar point at 
    // radarPointIdx
    Position pos = index_to_position(row, col, mapInfo, mapRel);
    
    float rPosx = radarData[array_index(radarPointIdx, 0, radarInfo)];
    float rPosy = radarData[array_index(radarPointIdx, 1, radarInfo)];
    // printf("rpos %d x: %f, y: %f\n", threadIdx.x, rPosx, rPosy);

    Position difference(
        pos.x - rPosx,
        pos.y - rPosy
    );
    return difference;
}

__device__ 
Position index_to_position(size_t row, size_t col, array_info *info, array_rel *relation){
    // find the position from center of map given cell index
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = (col - center_x);
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    Position ret(
        x_offset / (float)relation->res,
        y_offset / (float)relation->res
    );
    return ret;
}

__device__ __forceinline__
float calcPdf(float stdDev, float mean, float radius){
    // calculate the pdf of the given radar point based on the radius
    float variance = pow(stdDev, 2);
    float first = (1 / stdDev) * rsqrt(2 * HIP_PI_F);
    float exponent = -1 * pow(radius - mean, 2) / (2 * variance);
    float second = exp(exponent);
    return first*second;
}

__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      RadarData_t *radarData, 
                      array_info *mapInfo, 
                      array_rel* mapRel, 
                      array_info* radarInfo,
                      distInfo_t* distributionInfo,
                      radarId_t *radarIds){
    // In this function, the radar point id is threadIdx.x

    union{
        radarId_t radData;
        unsigned long long int ulong;
    } un;

    for(size_t col = 0; col < mapInfo->cols; col++){
        // find where the cell is relative to the radar point
        Position diff = indexDiff(blockIdx.x, col, 
                                    radarData, threadIdx.x, 
                                    radarInfo, mapInfo, mapRel);
        // don't calculate the pdf of this cell if it's too far away
        if(diff.radius > distributionInfo->distCutoff)
            continue;

        float pdfVal = calcPdf(distributionInfo->stdDev, distributionInfo->mean, diff.radius);
        // printf("pdf: %f\n", pdfVal);
        atomicAdd(&gaussMap[array_index(blockIdx.x,col,mapInfo)], pdfVal);

        un.radData.radarId = threadIdx.x;
        un.radData.probability = pdfVal;
        atomicMax((unsigned long long int*)&radarIds[array_index(blockIdx.x, col, mapInfo)], un.ulong);

    }
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed
    checkCudaError(hipMemcpy(radarInfo_cuda, &radarInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));

    // dispatch the kernel with `numPoints x mapInfo.rows` threads
    radarPointKernel<<<mapInfo.rows,radarInfo.rows>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c,
        radarIds
    );
    
    // wait untill all threads sync
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

}

//-----------------------------------------------------------------------------
// maxima locating

__global__
void calcMaxKernel(maxVal_t *isMax, 
                  float* array, array_info *mapInfo,
                  radarId_t *radarIds){
    int row = threadIdx.x;
    int col = blockIdx.x;
    if(row == 0 || row == mapInfo->rows) return;
    if(col == 0 || col == mapInfo->cols) return;
    
    float curVal = array[array_index(row,col, mapInfo)];
    if(curVal == 0) return; // not a max if it's zero

    maxVal_t *toInsert;
    toInsert = &isMax[array_index(row,col,mapInfo)];
    size_t iterator = 0;
    for(int i = -3; i <= 3; i++){
        for(int j = -3; j <= 3; j++){
            if(array[array_index(row+i, col+j, mapInfo)] > curVal)
                return;
            if(row+i >= 0 && col+j >= 0)
                toInsert->radars[iterator++] = radarIds[array_index(row+i, col+j, mapInfo)].radarId;
        }
    }

    toInsert->isMax = 1;
    toInsert->classVal = 0;
}

__device__ __forceinline__
float calcMean(size_t col, int16_t* radars, RadarData_t *radarData, array_info *radarInfo){
    float total = 0;
    size_t numPoints = 0;
    for(size_t i = 0; i < 49; i++){
        if(radars[i] == -1) continue;

        total += radarData[array_index(radars[i], col, radarInfo)];
        numPoints++;
    }
    
    return (total/numPoints);
}

__global__ 
void aggregateMax(mapType_t *array, array_info *mapInfo, array_rel *mapRel,
                             maxVal_t *isMax, float* ret, radarId_t *radarIds,
                             array_info* maxInfo, float minCutoff,
                             RadarData_t *radarData, array_info *radarInfo){
    // creates an array with the return information in the form of:
    // [row, col, class, pdfVal, vx, vy]
    size_t maxFound = 0;
    maxVal_t tmp;
    for(size_t row = 0; row < mapInfo->rows; row++){
        for(size_t col = 0; col < mapInfo->cols; col++){
            tmp = isMax[(size_t)(row * mapInfo->cols + col)];
            if(tmp.isMax == 1 && array[row * mapInfo->cols + col] >= minCutoff){
                if(maxFound++ == threadIdx.x){
                    ret[array_index(threadIdx.x, 0, maxInfo)] = ((float)(row - mapInfo->rows/2.0) * -1.0) / mapRel->res;
                    ret[array_index(threadIdx.x, 1, maxInfo)] = (col - mapInfo->cols/2.0) / mapRel->res;
                    ret[array_index(threadIdx.x, 2, maxInfo)] = radarData[array_index(tmp.radars[49/2],3, radarInfo)];
                    ret[array_index(threadIdx.x, 3, maxInfo)] = array[row * mapInfo->cols + col];
                    ret[array_index(threadIdx.x, 4, maxInfo)] = calcMean(8, tmp.radars, radarData, radarInfo);
                    ret[array_index(threadIdx.x, 5, maxInfo)] = calcMean(9, tmp.radars, radarData, radarInfo);
                }
            }
        }
    }
}

std::pair<array_info,float*> GaussMap::calcMax(){
    maxVal_t *isMax_cuda;
    checkCudaError(hipMalloc(&isMax_cuda, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols));

    // initialize isMax to 0
    checkCudaError(hipMemset(isMax_cuda, 0, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols));

    dim3 maxGridSize(mapInfo.rows, 1, 1);
    dim3 maxBlockSize(mapInfo.cols, 1, 1);

    calcMaxKernel<<<maxGridSize, maxBlockSize>>>(
        isMax_cuda,
        array,
        mapInfo_cuda,
        radarIds
    );

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    // copy back to host so we can iterate over it
    maxVal_t *isMax = (maxVal_t*)calloc(sizeof(maxVal_t), mapInfo.rows * mapInfo.cols);
    checkCudaError(hipMemcpy(isMax, isMax_cuda, sizeof(maxVal_t) * mapInfo.rows * mapInfo.cols, hipMemcpyDeviceToHost));
    
    float *arrayTmp = (float*)calloc(sizeof(float), mapInfo.rows * mapInfo.cols);
    checkCudaError(hipMemcpy(arrayTmp, array, sizeof(float) * mapInfo.rows * mapInfo.cols, hipMemcpyDeviceToHost));

    // find the number of maxima
    // this can be optimized later
    size_t numMax = 0;
    maxVal_t tmp;
    for(size_t row = 0; row < mapInfo.rows; row++){
        for(size_t col = 0; col < mapInfo.cols; col++){
            tmp = isMax[(size_t)(row * mapInfo.cols + col)];
            if(tmp.isMax == 1 && arrayTmp[row * mapInfo.cols + col] >= minCutoff){
                numMax++;
            }
        }
    }
    
    array_info maxData;
    maxData.cols = 6;
    maxData.rows = numMax;
    maxData.elementSize = sizeof(float);

   
    array_info *maxData_c;
    checkCudaError(hipMalloc(&maxData_c, sizeof(array_info)));
    checkCudaError(hipMemcpy(maxData_c, &maxData, sizeof(array_info), hipMemcpyHostToDevice));

    float *ret, *ret_c;
    checkCudaError(hipMalloc(&ret_c, sizeof(float) * maxData.rows * maxData.cols));

    aggregateMax<<<1, numMax>>>(
        array,
        mapInfo_cuda,
        mapRel_cuda,
        isMax_cuda,
        ret_c,
        radarIds,
        maxData_c,
        minCutoff,
        radarData,
        radarInfo_cuda
    );

    hipDeviceSynchronize();
    hipError_t error2 = hipGetLastError();
    if(error2 != hipSuccess){
        std::stringstream ss;
        ss << "aggregateMaxKernel launch failed\n";
        ss << hipGetErrorString(error2);
        throw std::runtime_error(ss.str());
    }

    ret = (float*)malloc(maxData.rows * maxData.cols * sizeof(float));
    checkCudaError(hipMemcpy(ret, ret_c, maxData.elementSize * maxData.rows * maxData.cols, hipMemcpyDeviceToHost));

    safeCudaFree(ret_c);
    safeCudaFree(isMax_cuda);
    safeCudaFree(maxData_c);

    return std::pair<array_info,float*>(maxData,ret);
}

//-----------------------------------------------------------------------------
// Position class implementation
__device__ 
Position::Position(float X, float Y) : x(X), y(Y){
    radius = hypotf(x,y);
}

__device__
Position::Position(){

}

__device__
void Position::recalc(){
    radius = hypotf(x,y);
}

//-----------------------------------------------------------------------------
