#include "hip/hip_runtime.h"
#include "derivative.cuh"
#include "gaussMap.cuh"

__global__
void calcDerivativeKernel(float* f, array_info *fInfo, float* fprime, array_info *fPrimeInfo){
    // https://en.wikipedia.org/wiki/Finite_difference#Multivariate_finite_differences 
    // ^ 5th equation

}

float* GaussMap::calcDerivative(){
    primeInfo.rows = mapInfo.rows -2;
    primeInfo.cols = mapInfo.cols -2;
    primeInfo.elementSize = sizeof(float);

    checkCudaError(hipMalloc(&primeInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMemcpy(primeInfo_cuda, &primeInfo, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    if(mapInfo_cuda == nullptr)
        throw std::runtime_error("radar data must be added before the derivative can be calculated.");
    
    checkCudaError(hipMalloc(&arrayPrime, sizeof(float) * primeInfo.rows * primeInfo.cols));

    // dispatch the kernel with a single thread per cell
    calcDerivativeKernel<<<primeInfo.rows, primeInfo.cols>>>(
        array,
        mapInfo_cuda,
        arrayPrime,
        primeInfo_cuda
    );

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "calcDerivativeKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::runtime_error(ss.str());
    }

    hipDeviceSynchronize();

    return nullptr;
}