#include "hip/hip_runtime.h"
#include <sstream>
#include "gaussMap.cuh"

__device__
size_t array_index(size_t row, size_t col, array_info *info){
    // helper function to find the array index
    return row * info->rows + col;
}

template <typename T>
__device__
double radiusFromPos(T x, T y){
    // return the radius from the position from origin (at center of map)
    // x and y in meters
    return hypot((double)x, (double)y);
}

__device__ 
dim3 index_to_position(size_t row, size_t col, array_info *info, array_rel *relation){
    // find the position from center of map given cell index
    // ret: dim3 (x,y,radius)
    float center_x = (float)(info->cols/2.0);
    float center_y = (float)(info->rows/2.0);
    float x_offset = col - center_x;
    float y_offset = (row - center_y) * -1;     // flip the y axis so + is in the direction of travel

    dim3 ret;
    ret.x = x_offset / relation->res;
    ret.y = y_offset / relation->res;
    ret.z = radiusFromPos(ret.x, ret.y);

    return ret;
}



__global__ 
void radarPointKernel(mapType_t* gaussMap, 
                      RadarData_t *radarData, 
                      array_info *mapInfo, 
                      array_rel* mapRel, 
                      array_info* radarInfo,
                      double* distributionInfo){
    // printf("here!\n");
}

void GaussMap::calcRadarMap(){

    // allocate this struct in shared memory so we don't have to copy
    // it to each kernel when it's needed
    array_info *tmpa, *tmpb;
    array_rel *tmpc;
    tmpa = (array_info*)malloc(sizeof(struct Array_Info));
    tmpb = (array_info*)malloc(sizeof(struct Array_Info));
    tmpc = (array_rel*)malloc(sizeof(struct Array_Relationship));
    memcpy(tmpa, &mapInfo, sizeof(struct Array_Info));
    memcpy(tmpb, &radarInfo, sizeof(struct Array_Info));
    memcpy(tmpc, &mapRel, sizeof(struct Array_Relationship));

    checkCudaError(hipMalloc(&mapInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&radarInfo_cuda, sizeof(struct Array_Info)));
    checkCudaError(hipMalloc(&mapRel_cuda, sizeof(struct Array_Relationship)));
    checkCudaError(hipMalloc(&radarDistri_c, 2*sizeof(double)));
    checkCudaError(hipMemcpy(mapInfo_cuda, tmpa, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarInfo_cuda, tmpb, sizeof(struct Array_Info), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(mapRel_cuda, tmpc, sizeof(struct Array_Relationship), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(radarDistri_c, radarDistri, 2*sizeof(double), hipMemcpyHostToDevice));

    free(tmpa);
    free(tmpb);
    free(tmpc);


    // dispatch the kernel with `numPoints` threads
    radarPointKernel<<<1,numPoints>>>(
        array,
        radarData,
        mapInfo_cuda,
        mapRel_cuda,
        radarInfo_cuda,
        radarDistri_c
    );

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        std::stringstream ss;
        ss << "radarPointKernel launch failed\n";
        ss << hipGetErrorString(error);
        throw std::string(ss.str());
    }

    // wait untill all threads sync
    hipDeviceSynchronize();
}
