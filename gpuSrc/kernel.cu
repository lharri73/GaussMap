#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void kernel
(double *vec, double scalar, int num_elements)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    vec[idx] = vec[idx] * scalar;
  }
}


void run_kernel
(double *vec, double scalar, int num_elements)
{
  dim3 dimBlock(256, 1, 1);
  dim3 dimGrid(ceil((double)num_elements / dimBlock.x));
  
  kernel<<<dimGrid, dimBlock>>>
    (vec, scalar, num_elements);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw strstr.str();
  }
}
